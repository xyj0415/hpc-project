#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdint.h>
#include <algorithm>
#include <omp.h>

#define BLOCK_SIZE 32
typedef unsigned char ubyte;
typedef unsigned int uint;

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

void init_world(uint8_t* world, int width, int height){
	for(int i = 0; i< width*height; i++)
		world[i] = rand()%2;
}

void print_world(uint8_t* world, int width, int height){
	for(int i = 0; i< width; i++){
		for(int j =0; j < height; j++)
			printf("%d ", world[i*width+j]);
		printf("\n");
	}
}

__global__ void game_kernel(ubyte* current, ubyte* next, uint width, uint height, uint board_size){
	uint x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	uint y = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	uint index = y*width+x;
	uint x_left = y*width + ((x+width-1)&(width-1));
	uint x_right = y*width + ((x+1)&(width-1));
	uint y_up = ((y+width-1)&(width-1))*width+x;
	uint y_down = ((y+1)&(width-1))*width+x;
	uint i_ne = ((y+width-1)&(width-1))*width+((x+1)&(width-1));
	uint i_nw = ((y+width-1)&(width-1))*width+((x+width-1)&(width-1));
	uint i_se = ((y+1)&(width-1))*width+((x+1)&(width-1));
	uint i_sw = ((y+1)&(width-1))*width+((x+width-1)&(width-1));

	int alive_cell = current[x_left]+current[x_right]+current[y_up]+current[y_down]+current[i_ne]+current[i_nw]+current[i_se]+current[i_sw];
	//printf("%d, %d", index, alive_cell);
	//int alive_cell = current[x-1*width+y]+current[x+1*width+y]+current[x*width+y-1]+current[x*width+y+1]+current[x-1*width+y-1]+current[x-1*width+y+1]+current[x+1*width+y-1]+current[x+1*width+y+1];
	next[index] = alive_cell == 3 || (alive_cell ==2 && current[index]) ? 1:0;

}	

void run_game(ubyte* world, ubyte* current, ubyte* next, uint width, uint height, int iteration){
	dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid_dim(width/BLOCK_SIZE, height/BLOCK_SIZE);
	printf("%d, %d\n", width/BLOCK_SIZE, height/BLOCK_SIZE);

	for(int i=0; i< iteration; i++){
		//printf("%d\n", i);
		game_kernel<<<grid_dim, block_dim>>>(current, next, width, height, width*height);
/*		ubyte *temp = current;
		current = next;
		next = temp;*/

		std::swap(current, next);
		Check_CUDA_Error("test");
		//printf("itr: %d\n", i);
		//cudaMemcpyAsync(world, current, width * height * sizeof(ubyte), cudaMemcpyDeviceToHost);
		//cudaDeviceSynchronize();
		//print_world(world, width, height);
	}
}


int main(int argc, char const *argv[]){
	/* code */
	uint width = 1024;
	uint height = 1024;
	int iteration = 100000;
	srand(time(NULL));

	ubyte *world;
	hipHostMalloc((void**)&world, width * height * sizeof(ubyte), hipHostMallocDefault);
	init_world(world, width, height);
	//print_world(world, width, height);

	ubyte *g_world_current, *g_world_next;
	hipMalloc(&g_world_current, width * height * sizeof(ubyte));
	hipMalloc(&g_world_next, width * height * sizeof(ubyte));
	hipMemcpyAsync(g_world_current, world, width * height * sizeof(ubyte), hipMemcpyHostToDevice);
	hipMemcpyAsync(g_world_next, world, width * height * sizeof(ubyte), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	double tt = omp_get_wtime();
	run_game(world, g_world_current, g_world_next, width, height, iteration);
	printf("time: %lf\n", (omp_get_wtime()-tt));
	hipFree(g_world_current);
	hipFree(g_world_next);
	hipFree(world);
	return 0;
}